#include "hip/hip_runtime.h"
#include "config.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include "cuda_fake.h"

PT_KERNEL void kernelMix(hipSurfaceObject_t surfaceNew, hipSurfaceObject_t surfaceAcc, int width, int height, unsigned int sampleCount) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int byteOffset = x * sizeof(float4);

    float4 newSample, accSample;
    surf2Dread(&newSample, surfaceNew, byteOffset, y);
    surf2Dread(&accSample, surfaceAcc, byteOffset, y);

    float inv_sampleCount = 1.0f / sampleCount;

    accSample.x = accSample.x * (1.0f - inv_sampleCount) + newSample.x * inv_sampleCount;
    accSample.y = accSample.y * (1.0f - inv_sampleCount) + newSample.y * inv_sampleCount;
    accSample.z = accSample.z * (1.0f - inv_sampleCount) + newSample.z * inv_sampleCount;
    accSample.w = accSample.w * (1.0f - inv_sampleCount) + newSample.w * inv_sampleCount;

    surf2Dwrite(accSample, surfaceAcc, byteOffset, y);
}

extern "C" void launchMixKernel(hipSurfaceObject_t surfaceNew, hipSurfaceObject_t surfaceAcc, int width, int height, unsigned int sampleCount) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    kernelMix<<<gridSize, blockSize>>>(surfaceNew, surfaceAcc, width, height, sampleCount);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) std::cerr << "CUDA Kernel failed: " << hipGetErrorString(err) << std::endl;
}