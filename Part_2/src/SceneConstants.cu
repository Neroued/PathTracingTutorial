#include "config.h"

#include "SceneConstants.cuh"
#include <hip/hip_runtime.h>
#include <cstdio>


BEGIN_NAMESPACE_PT

__constant__ SceneConstants d_sceneConstants;

PT_CPU void uploadSceneConstant(const SceneConstants& host) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_sceneConstants), &host, sizeof(pt::SceneConstants));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) { fprintf(stderr, "hipMemcpyToSymbol failed: %s\n", hipGetErrorString(err)); }
}

END_NAMESPACE_PT