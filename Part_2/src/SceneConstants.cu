#include "config.h"

#include "SceneConstants.cuh"
#include <hip/hip_runtime.h>
#include <cstdio>


BEGIN_NAMESPACE_PT

__constant__ SceneConstants d_sceneConstants;

PT_CPU void uploadSceneConstant(const SceneConstants& host) {
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_sceneConstants), &host, sizeof(pt::SceneConstants)));
}

END_NAMESPACE_PT